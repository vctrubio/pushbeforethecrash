#include "hip/hip_runtime.h"
#ifndef VIGNETTE_CU
    #define VIGNETTE_CU

    #include "VignetteGPU.h"

    #if __CUDACC_VER_MAJOR__ >= 9
        #include <hip/hip_fp16.h>
    #endif

	struct Pixel16 {
		unsigned short x;
		unsigned short y;
		unsigned short z;
		unsigned short w;
	};
	inline __device__ float4 HalfToFloat4 ( Pixel16 in )
	{
		return make_float4 ( __half2float(in.x), __half2float(in.y), __half2float(in.z), __half2float(in.w ) );
	}
	inline __device__ Pixel16 FloatToHalf4 ( float4 in )
	{
		Pixel16 v;
		v.x = __float2half_rn( in.x ); v.y = __float2half_rn( in.y ); v.z = __float2half_rn( in.z ); v.w = __float2half_rn( in.w );
		return v;
	}

	// Vignette CUDA kernel

	__global__ void kVignetteCUDA (
		float4* destImg,
		const int destPitch,
		const int in16f,
		const int inWidth,
		const int inHeight,
		const float	amountF,
		const float	outer_aaF,
		const float	outer_bbF,
		const float	outer_abF,
		const float	inner_aaF,
		const float	inner_bbF,
		const float	inner_aabbF,
		const float	inner_abF,
		float	x_t,
		float	y_t
		)	
	{
		float4 dest;

		int x = blockIdx.x*blockDim.x + threadIdx.x;
		int y = blockIdx.y*blockDim.y + threadIdx.y;

		if ( x >= inWidth || y >= inHeight ) return;

		if ( in16f ) {
			Pixel16*  in16 = (Pixel16*)  destImg;			
			dest =  HalfToFloat4 (  in16[y *  destPitch + x] );
		} else {
			dest =  destImg[y *  destPitch + x];
		}
		
		x_t = x - x_t;
		y_t = y - y_t;

		float f = 0.0f;

		if (x_t != 0 || y_t != 0)
		{

			float xx = x_t * x_t;
			float yy = y_t * y_t;

			if (inner_aaF == outer_aaF)
			{
				f = xx*inner_bbF + yy * inner_aaF < inner_aabbF ? 0.0f : 1.0f;
			}
			else {
				float R = sqrt(xx + yy),
					r_i = inner_abF / sqrt(inner_bbF * xx + inner_aaF * yy) * R,
					r_o = outer_abF / sqrt(outer_bbF * xx + outer_aaF * yy) * R;
				f = min(1.0f, max(0.0f, (R - r_i) / (r_o - r_i)));
			}
		}

		f *= amountF;

		dest.x = max(0.0f,dest.x + f);
		dest.y = max(0.0f, dest.y + f);
		dest.z = max(0.0f, dest.z + f);

		if ( in16f ) {
			Pixel16*  dest16 = (Pixel16*) destImg;
			dest16[y * destPitch + x] = FloatToHalf4 ( dest );
		} else {
			destImg[y * destPitch + x] = dest;
		}
	}
	
	void Vignette_CUDA (
		float *destBuf,
		int destPitch,
		int	is16f,
		int width,
		int height,
		VigInfoGPU *viP )
	{
		dim3 blockDim (16, 16, 1);
		dim3 gridDim ( (width + blockDim.x - 1)/ blockDim.x, (height + blockDim.y - 1) / blockDim.y, 1 );		

		kVignetteCUDA <<< gridDim, blockDim, 0 >>> ( (float4*) destBuf, destPitch, is16f, width, height,  
			viP->amountF,
			viP->outer_aaF,
			viP->outer_bbF,
			viP->outer_abF,
			viP->inner_aaF,
			viP->inner_bbF,
			viP->inner_aabbF,
			viP->inner_abF,
			viP->x_t,
			viP->y_t);

		hipDeviceSynchronize();
	}

#endif 
