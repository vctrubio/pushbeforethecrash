#include "hip/hip_runtime.h"

#ifndef SDK_CROSS_DISSOLVE
	#define SDK_CROSS_DISSOLVE

    #if __CUDACC_VER_MAJOR__ >= 9
        #include <hip/hip_fp16.h>
    #endif
	#include "PrGPU/KernelSupport/KernelCore.h" //includes KernelWrapper.h
	#include "PrGPU/KernelSupport/KernelMemory.h"

	#if GF_DEVICE_TARGET_DEVICE
		GF_KERNEL_FUNCTION(kCrossDissolveCUDA,
			((GF_PTR(float4 const))(outImg))		
			((GF_PTR(float4 const))(inImg))
			((GF_PTR(float4))(destImg)),
			((unsigned int)(outPitch))
			((unsigned int)(inPitch))
			((unsigned int)(destPitch))
			((int)(in16f))
			((unsigned int)(inWidth))
			((unsigned int)(inHeight))
			((float)(inProgress))
			((int)(inFlip)),
			((uint2)(inXY)(KERNEL_XY)))
		{
			float4 outgoing, incoming, dest;

			if ( inXY.x >= inWidth || inXY.y >= inHeight ) return;

			outgoing = ReadFloat4(outImg, inXY.y * outPitch + inXY.x, !!in16f);
			incoming = ReadFloat4(inImg, inXY.y * inPitch + inXY.x, !!in16f);
		
			float outgoingAlphaWeighted = outgoing.w * (1.0f - inProgress);
			float incomingAlphaWeighted  = incoming.w * inProgress; 
			float newAlpha = outgoingAlphaWeighted  + incomingAlphaWeighted ; 
			float recipNewAlpha = newAlpha != 0.0f ? 1.0f / newAlpha : 0.0f;

			dest.x = (outgoing.x * outgoingAlphaWeighted + incoming.x * incomingAlphaWeighted) * recipNewAlpha; 
			dest.y = (outgoing.y * outgoingAlphaWeighted + incoming.y * incomingAlphaWeighted) * recipNewAlpha; 
			dest.z = (outgoing.z * outgoingAlphaWeighted + incoming.z * incomingAlphaWeighted) * recipNewAlpha; 
			dest.w = newAlpha;

			WriteFloat4(dest, destImg, inXY.y * outPitch + inXY.x, !!in16f);	
		}
	#endif

	#if __NVCC__
		void CrossDissolve_CUDA (
			float const *outBuf,		
			float const *inBuf,
			float *destBuf,
			unsigned int outPitch,
			unsigned int inPitch,
			unsigned int destPitch,
			int	is16f,
			unsigned int width,
			unsigned int height,
			float progress,
			int flip )
		{
			dim3 blockDim (16, 16, 1);
			dim3 gridDim ( (width + blockDim.x - 1)/ blockDim.x, (height + blockDim.y - 1) / blockDim.y, 1 );		

			kCrossDissolveCUDA <<< gridDim, blockDim, 0 >>> ( (float4 const*) outBuf, (float4 const*) inBuf, (float4*) destBuf, outPitch, inPitch, destPitch, is16f, width, height, progress, flip );

			hipDeviceSynchronize();
		}
	#endif //GF_DEVICE_TARGET_HOST

#endif //SDK_CROSS_DISSOLVE
